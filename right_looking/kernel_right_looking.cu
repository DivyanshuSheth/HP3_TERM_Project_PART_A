#include "right_looking.cu"

float * copyDataToDevice (float *d_data, float *h_data, int totalElements) {

    hipError_t err = hipSuccess;
    err = hipMalloc((void **)&d_data, totalElements);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector d_data (error code %s)!\n",
             hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copying data from host memory to the CUDA device\n");
    err = hipMemcpy(d_data, h_data, totalElements, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy h_data from host to device (error code %s)\n", 
             hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    printf("Copied data successfully!\n");
    
    return d_data;
}

int main()
{
    FILE *fptr;
    fptr = fopen("input.txt", "r");
    int n, dim;
    // char str[50];
    int temp;
    fscanf(fptr, "%d", &n);
    fscanf(fptr, "%d", &dim);
    float h_A[n*dim*dim];
    int count = 0;
    int x = 0;
    int gidx = 0;

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < dim; j++)
        {
            for (int k = 0; k < dim; k++)
            {
                fscanf(fptr, "%d", &temp);
                x = j * dim + k;
                gidx = x * n + i;
                h_A[gidx] = temp;
            }
        }
    }
    int size = n * dim * dim;
    
    float *d_A = NULL;
    float * read_data = copyDataToDevice(d_A, h_A, size);
    int N = dim, i, j;
//     float M[n*dim*dim] = h_A;
    printf("Testing for matrix M [%dx%d]\n",N,N);
    dim3 grid(1,1,1);
    dim3 block(TILE_SIZE,TILE_SIZE,1);
    right_looking_launch_kernel<<<grid,block>>>(read_data,N);
    hipError_t err = hipMemcpy(h_A,read_data,size,hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy the output matrix M from device to Host (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("Printing output matrix\n");
    for(i=0;i<n;i++)
    {
        for(j=0;j<n;j++)
        {
            if(j<=i)
                printf("%f\t",h_A[i*N + j]);
            else
                printf("%f\t",0.0);
        }
        printf("\n");
    }
    err = hipFree(read_data);
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix M (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipDeviceReset();
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the CUDA device (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    free(h_A);
    printf("DONE!\n");
    return 0;
}
